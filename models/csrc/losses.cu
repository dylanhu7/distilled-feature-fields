#include "hip/hip_runtime.h"
#include "utils.h"
#include <thrust/scan.h>


// for details of the formulae, please see https://arxiv.org/pdf/2206.05085.pdf

template <typename scalar_t>
__global__ void prefix_sums_kernel(
    const scalar_t* ws,
    const scalar_t* wts,
    const torch::PackedTensorAccessor64<int64_t, 2, torch::RestrictPtrTraits> rays_a,
    scalar_t* ws_prefix_sum,
    scalar_t* wts_prefix_sum
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= rays_a.size(0)) return;

    const int start_idx = rays_a[n][1], N_samples = rays_a[n][2];

    // compute prefix sum of ws and ws*ts
    // [a0, a1, a2, a3, ...] -> [0, a0, a0+a1, a0+a1+a2, ...]
    thrust::exclusive_scan(thrust::device,
                           ws+start_idx,
                           ws+start_idx+N_samples,
                           ws_prefix_sum+start_idx);
    thrust::exclusive_scan(thrust::device,
                           wts+start_idx,
                           wts+start_idx+N_samples,
                           wts_prefix_sum+start_idx);
}


template <typename scalar_t>
__global__ void reduce_distortion_loss_kernel(
    const scalar_t* _loss,
    const torch::PackedTensorAccessor64<int64_t, 2, torch::RestrictPtrTraits> rays_a,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> loss
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= rays_a.size(0)) return;

    const int ray_idx = rays_a[n][0], start_idx = rays_a[n][1], N_samples = rays_a[n][2];

    loss[ray_idx] = thrust::reduce(thrust::device, 
                                   _loss+start_idx,
                                   _loss+start_idx+N_samples,
                                   (scalar_t)0);
}


std::vector<torch::Tensor> distortion_loss_fw_cu(
    const torch::Tensor ws,
    const torch::Tensor deltas,
    const torch::Tensor ts,
    const torch::Tensor rays_a
){
    const int N_rays = rays_a.size(0), N = ws.size(0);

    auto wts = ws * ts;

    auto ws_prefix_sum = torch::zeros({N}, ws.options());
    auto wts_prefix_sum = torch::zeros({N}, ws.options());

    const int threads = 256, blocks = (N_rays+threads-1)/threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(ws.type(), "distortion_loss_fw_cu_prefix_sum", 
    ([&] {
        prefix_sums_kernel<scalar_t><<<blocks, threads>>>(
            ws.data_ptr<scalar_t>(),
            wts.data_ptr<scalar_t>(),
            rays_a.packed_accessor64<int64_t, 2, torch::RestrictPtrTraits>(),
            ws_prefix_sum.data_ptr<scalar_t>(),
            wts_prefix_sum.data_ptr<scalar_t>()
        );
    }));

    auto _loss = 2*ws*(ts*ws_prefix_sum-wts_prefix_sum) + 1.0f/3*ws*ws*deltas;

    auto loss = torch::zeros({N_rays}, ws.options());

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(ws.type(), "distortion_loss_fw_cu_reduce_loss", 
    ([&] {
        reduce_distortion_loss_kernel<scalar_t><<<blocks, threads>>>(
            _loss.data_ptr<scalar_t>(),
            rays_a.packed_accessor64<int64_t, 2, torch::RestrictPtrTraits>(),
            loss.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return {loss, ws_prefix_sum, wts_prefix_sum};
}


template <typename scalar_t>
__global__ void distortion_loss_bw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> dL_dloss,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ws_prefix_sum,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> wts_prefix_sum,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ws,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> deltas,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ts,
    const torch::PackedTensorAccessor64<int64_t, 2, torch::RestrictPtrTraits> rays_a,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> dL_dws
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= rays_a.size(0)) return;

    const int ray_idx = rays_a[n][0], start_idx = rays_a[n][1], N_samples = rays_a[n][2];
    const int end_idx = start_idx+N_samples-1;

    const scalar_t ws_total = ws_prefix_sum[end_idx]+ws[end_idx];
    const scalar_t wts_total = wts_prefix_sum[end_idx]+ws[end_idx]*ts[end_idx];
    // fill in dL_dws from start_idx to start_idx+N_samples-1
    for (int s=start_idx; s<=end_idx; s++){
        dL_dws[s] = dL_dloss[ray_idx] * 2 * (
            ts[s]*ws_prefix_sum[s] - wts_prefix_sum[s] +
            (s==end_idx?
                (scalar_t)0:
                (wts_total-wts_prefix_sum[s+1]-ts[s]*(ws_total-ws_prefix_sum[s+1]))
            )
        );
        dL_dws[s] += dL_dloss[ray_idx] * (scalar_t)2/3*ws[s]*deltas[s];
    }
}


torch::Tensor distortion_loss_bw_cu(
    const torch::Tensor dL_dloss,
    const torch::Tensor ws_prefix_sum,
    const torch::Tensor wts_prefix_sum,
    const torch::Tensor ws,
    const torch::Tensor deltas,
    const torch::Tensor ts,
    const torch::Tensor rays_a
){
    const int N_rays = rays_a.size(0), N = ws.size(0);

    auto dL_dws = torch::zeros({N}, dL_dloss.options());

    const int threads = 256, blocks = (N_rays+threads-1)/threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(ws.type(), "distortion_loss_bw_cu", 
    ([&] {
        distortion_loss_bw_kernel<scalar_t><<<blocks, threads>>>(
            dL_dloss.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ws_prefix_sum.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            wts_prefix_sum.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ws.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            deltas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ts.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rays_a.packed_accessor64<int64_t, 2, torch::RestrictPtrTraits>(),
            dL_dws.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return dL_dws;
}